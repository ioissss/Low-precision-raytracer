#include "utils/exception.hpp"

#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

namespace rt {
void check_cuda_error(hipError_t code) {
    if (code != hipSuccess) {
        throw std::exception(hipGetErrorString(code));
    }
}
} // namespace rt